#include "hip/hip_runtime.h"
#include <stdio.h>
#include <glob.h>
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>

#include "jkTools.h"

using namespace std;

/* ============================================================================
 * GPU functions
 * =========================================================================*/
/*__global__ void add(int *a, int *b, int *c){
    int tid = threadIdx.x;       //handle data at this index
    if (tid < N){
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void average(double *a, double *b, double *c){
    int tid = threadIdx.x;       //handle data at this index
    if (tid < N){
        c[tid] = a[tid] + b[tid];
    }
}*/

/* ============================================================================
 * Begin Main.
 * =========================================================================*/
int main(void){

    JackKnifeTools jk;
 
    /* check if data file is empty before writing */
    ifstream filestr("JackknifeData.dat");
    if(!jk.IsEmpty(filestr)){
        cout<<"\nError: Data file is not empty!\n"<<endl;
        return 0;
    }

    /* open file to write analyzed data to disk */
    FILE *file;
    file = fopen("JackknifeData.dat", "a+");
    fprintf(file,"#%9s\t%10s\t%10s\n", "T", "Cv", "CvErr");
    
    /* vector of estimator file names */
    vector<string> estFiles = jk.Glob("*estimator*");

    /* loop over all estimator files */
    for (int i = 0; i<estFiles.size(); i++){
        cout<<estFiles[i]<<endl;
        string temp = estFiles[i].substr(13,6);
        
        /* read in file */
        ifstream inFile;
        inFile.open(estFiles[i].c_str());

        /* grab whitespace delimited data */
        vec_dVec allData = jk.ReadData(inFile);
       
        /* bookkeeping */
        const int nBin = allData.size();
        
        /* store specific heat data */
        thrust::host_vector<double> Cv1(nBin), Cv2(nBin), Cv3(nBin);
        for (int j=0; j<nBin; j++){
            Cv1[j] = allData[j][11];
            Cv2[j] = allData[j][12];
            Cv3[j] = allData[j][13];
        }
        
        /* delete all unused data from memory */
        vector<dVec>().swap(allData);

        double jkTerm1, jkTerm2, jkTerm3, jkTermTot;
        double jkAve=0.0;       // jackknife average
        double jkAveSq=0.0;
        double rawAve1=0.0, rawAve2=0.0, rawAve3=0.0;

        thrust::host_vector<double> tempJKvec1(nBin);
        thrust::host_vector<double> tempJKvec2(nBin);
        thrust::host_vector<double> tempJKvec3(nBin);
        
        /* compute jackknife averages */
        for (int j=0; j<nBin; j++){
            /* get rid of one term from each vector */
            tempJKvec1 = Cv1, tempJKvec2 = Cv2, tempJKvec3 = Cv3;
            tempJKvec1.erase(tempJKvec1.begin() + j);
            tempJKvec2.erase(tempJKvec2.begin() + j);
            tempJKvec3.erase(tempJKvec3.begin() + j);
            
            /* compute average of subset of data */
            jkTerm1 = 0.0, jkTerm2 = 0.0, jkTerm3 = 0.0, jkTermTot = 0.0;
            for (int l=0; l<nBin-1; l++){
                jkTerm1 += tempJKvec1[l]/(1.0*(nBin-1));
                jkTerm2 += tempJKvec2[l]/(1.0*(nBin-1));
                jkTerm3 += tempJKvec3[l]/(1.0*(nBin-1));
            }
            jkTermTot = jkTerm1 - jkTerm2*jkTerm2 - jkTerm3;

            rawAve1 += Cv1[j]/(1.0*nBin);
            rawAve2 += Cv2[j]/(1.0*nBin);
            rawAve3 += Cv3[j]/(1.0*nBin);

            /* update running average */
            jkAve += jkTermTot/(1.0*nBin);
            jkAveSq += jkTermTot*jkTermTot/(1.0*nBin);
        }
        double rawAve = rawAve1 - rawAve2*rawAve2 - rawAve3;

        double actAve = 1.0*nBin*rawAve - 1.0*(nBin-1)*jkAve;
        double var = jkAveSq - jkAve*jkAve;
        double err = sqrt(1.0*(nBin-1)*var);

        cout<<"est = "<<actAve<<" +/- "<<err<<endl;

        /* temp stuff for writing */
        //double Cv = 0.0;
        //double CvErr = 0.0;

        /* create device arrays */
        //thrust::device_vector<double> Cv1_dev = Cv1;
        //thrust::device_vector<double> Cv2_dev = Cv1;
        //thrust::device_vector<double> Cv3_dev = Cv1;
        //thrust::device_vector<double> JkTerms_dev(nBin);
     
        /* allocate memory of GPU */
        //hipMalloc((void**)&dev_Cv1, nBin*sizeof(double));
        //hipMalloc((void**)&dev_b, sizeof(int));
        //hipMalloc((void**)&dev_c, sizeof(int));
        
        // copy 'a' and 'b' to GPU
        // HANDLE_ERROR( hipMemcpy(dev_a,a,N*sizeof(int),
        //            hipMemcpyHostToDevice));
        //hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
        // HANDLE_ERROR( hipMemcpy(dev_b,b,N*sizeof(int),
        //            hipMemcpyHostToDevice));
        //hipMemcpy(dev_b,b,N*sizeof(int), hipMemcpyHostToDevice);
        
        //add<<<200,N/200>>>(dev_a,dev_b,dev_c);

        // copy array 'c' back from GPU to CPU
        // HANDLE_ERROR( hipMemcpy(c,dev_c,N*sizeof(int),
        //            hipMemcpyDeviceToHost));
        //hipMemcpy(c,dev_c,N*sizeof(int), hipMemcpyDeviceToHost);

        // free memory allocated on GPU
        //hipFree(dev_a);
        //hipFree(dev_b);
        //hipFree(dev_c);

        //cout<<temp<<endl;
        fprintf(file,"%10s\t%10f\t%10f\n", temp.c_str(), actAve, err);
    }

    fclose(file);
    return 0;
}
